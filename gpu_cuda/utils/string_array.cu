#include "panic.cuh"
#include "string_array.cuh"
#include <cstddef>
#include <iostream>
#include <memory>
#include <tuple>

namespace rr::gpu_cuda::utils {
string_array create_string_array(size_t size, size_t total_length,
                                 size_t max_length) {
  string_array arr;
  arr.size = size;
  arr.total_length = total_length;
  arr.max_length = max_length;
  checkCudaErrors(hipMalloc(&arr.data, total_length * sizeof(char)));
  checkCudaErrors(hipMalloc(&arr.lengths, size * sizeof(size_t)));
  checkCudaErrors(hipMalloc(&arr.offsets, size * sizeof(size_t)));
  return arr;
}

string_array to_device_string_array(std::vector<std::string>::iterator begin,
                                    std::vector<std::string>::iterator end) {
  size_t size = std::distance(begin, end);

  std::vector<int> lengths(size);
  std::vector<int> offsets(size);
  size_t max_length = 0;
  size_t total_length = 0;

  for (auto it = begin; it != end; ++it) {
    size_t length = it->length();
    if (length > max_length) {
      max_length = length;
    }
    lengths[it - begin] = length;
    offsets[it - begin] = total_length;
    total_length += length;
  }

  auto data = std::make_unique<char[]>(total_length);
  for (auto it = begin; it != end; ++it) {
    std::copy(it->begin(), it->end(), data.get() + offsets[it - begin]);
  }

  auto d_arr = create_string_array(size, total_length, max_length);
  checkCudaErrors(hipMemcpy(d_arr.lengths, lengths.data(),
                             size * sizeof(size_t), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_arr.offsets, offsets.data(),
                             size * sizeof(size_t), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_arr.data, data.get(),
                             total_length * sizeof(char),
                             hipMemcpyHostToDevice));
  return d_arr;
}

void copy_string_array_to_vector(string_array &arr,
                                 std::vector<std::string>::iterator begin,
                                 std::vector<std::string>::iterator end) {}

void free_string_array(string_array &arr) {
  checkCudaErrors(hipFree(arr.data));
  checkCudaErrors(hipFree(arr.lengths));
  checkCudaErrors(hipFree(arr.offsets));
};
} // namespace rr::gpu_cuda::utils