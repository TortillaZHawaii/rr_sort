#include "hip/hip_runtime.h"
#include "enumeration_sort.cuh"
#include "utils/string_array.cuh"

namespace rr::gpu_cuda {
using namespace rr::gpu_cuda::utils;

__global__ void enumeration_sort_kernel(string_array input,
                                        string_array result) {
  auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= input.size)
    return;

  size_t length = input.lengths[tid];
  size_t offset = input.offsets[tid];

  int rank = 0;

  for (int i = 0; i < input.size; ++i) {
    if (i == tid)
      continue;

    size_t other_length = input.lengths[i];
    size_t other_offset = input.offsets[i];

    // compare strings
    int cmp = 0;
  }
}

void enumeration_sort(std::vector<std::string>::iterator begin,
                      std::vector<std::string>::iterator end) {
  auto d_arr = to_device_string_array(begin, end);
  auto d_res =
      create_string_array(d_arr.size, d_arr.max_length, d_arr.max_length);

  const uint grid_size = 1024;
  const uint block_size =
      d_arr.size / grid_size + (d_arr.size % grid_size) ? 1 : 0;

  enumeration_sort_kernel<<<grid_size, block_size>>>(d_arr, d_res);

  copy_string_array_to_vector(d_res, begin, end);

  free_string_array(d_arr);
  free_string_array(d_res);
}
} // namespace rr::gpu_cuda