#include "hip/hip_runtime.h"
#include "enumeration_sort.cuh"
#include "utils/panic.cuh"
#include "utils/string_array.cuh"
#include <cstddef>
#include <iostream>
#include <memory>

namespace rr::gpu_cuda {
using namespace rr::gpu_cuda::utils;

__device__ int compare_strings(const char *str1, const char *str2,
                               size_t length1, size_t length2) {
  int cmp = 0;
  for (int i = 0; i < length1 && i < length2; ++i) {
    if (str1[i] < str2[i]) {
      cmp = -1;
      break;
    } else if (str1[i] > str2[i]) {
      cmp = 1;
      break;
    }
  }

  if (cmp == 0) {
    if (length1 < length2)
      cmp = -1;
    else if (length1 > length2)
      cmp = 1;
  }

  return cmp;
}

__global__ void enumeration_sort_kernel(string_array input,
                                        size_t *result_positions) {
  auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= input.size)
    return;

  size_t length = input.lengths[tid];
  size_t offset = input.offsets[tid];

  size_t rank = 0;

  for (int i = 0; i < input.size; ++i) {
    if (i == tid)
      continue;

    size_t other_length = input.lengths[i];
    size_t other_offset = input.offsets[i];

    int cmp = compare_strings(input.data + offset, input.data + other_offset,
                              length, other_length);

    if (cmp > 0 || (cmp == 0 && tid > i)) {
      ++rank;
    }
  }

  // update result
  result_positions[tid] = rank;
}

void enumeration_sort(std::vector<std::string>::iterator begin,
                      std::vector<std::string>::iterator end) {
  auto d_arr = to_device_string_array(begin, end);
  size_t *d_res;
  checkCudaErrors(hipMalloc(&d_res, d_arr.size * sizeof(size_t)));
  // For some reason, lower grid count per block cannot be run on my machine on
  // bible.txt. 1024 is also max number for any CUDA device.
  //
  // https://stackoverflow.com/questions/6048907/maximum-blocks-per-gridcuda
  const uint grid_count_per_block = 1024;
  const uint block_count = d_arr.size / grid_count_per_block +
                           ((d_arr.size % grid_count_per_block) ? 1 : 0);
  enumeration_sort_kernel<<<grid_count_per_block, block_count>>>(d_arr, d_res);
  checkCudaErrors(hipPeekAtLastError());

  auto res = std::make_unique<size_t[]>(d_arr.size);
  checkCudaErrors(hipMemcpy(res.get(), d_res, d_arr.size * sizeof(size_t),
                             hipMemcpyDeviceToHost));

  std::vector<std::string> sorted(d_arr.size);
  for (int i = 0; i < d_arr.size; ++i) {
    sorted[res[i]] = *(begin + i);
  }

  for (int i = 0; i < d_arr.size; ++i) {
    *(begin + i) = std::move(sorted[i]);
  }

  checkCudaErrors(hipFree(d_res));
  free_string_array(d_arr);
}
} // namespace rr::gpu_cuda